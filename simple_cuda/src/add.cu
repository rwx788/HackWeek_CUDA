
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    y[i] = x[i] + y[i];
  }
}
// Kernel function to initialize the elements of two arrays
__global__ void init(int n, float *x, float *y) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
}

int main(void) {
  int N = 1 << 25;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));
  // Get device id
  int device = -1;
  hipGetDevice(&device);
  // Get number of Streaming Multiprocessors
  int numSMs;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, device);
  // initialize x and y arrays on the GPU
  init<<<32 * numSMs, 256>>>(N, x, y);

  // Run kernel on all elements on the GPU
  add<<<32 * numSMs, 256>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
